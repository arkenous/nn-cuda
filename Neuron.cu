#include "hip/hip_runtime.h"

#include <iostream>
#include <sstream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/functional.h>

#include "Neuron.cuh"

using std::vector;
using std::random_device;
using std::mt19937;
using std::uniform_real_distribution;
using std::max;
using std::string;
using std::stringstream;


struct learn_m_functor {
  double delta;
  double beta_one;

  learn_m_functor(double _delta, double _beta_one) {
    delta = _delta;
    beta_one = _beta_one;
  }

  __host__ __device__ double operator()(const double& m, const double& inputValue) const {
    return beta_one * m + (1 - beta_one) * (delta * inputValue);
  }
};


struct learn_nu_functor {
  double delta;
  double beta_two;

  learn_nu_functor(double _delta, double _beta_two) {
    delta = _delta;
    beta_two = _beta_two;
  }

  __host__ __device__ double operator()(const double& nu, const double& inputValue) const {
    return beta_two * nu + (1 - beta_two) * pow((delta * inputValue), 2);
  }
};


struct learn_functor {
  double beta_one;
  double beta_two;
  unsigned long iteration;
  double epsilon;
  double alpha;

  learn_functor(double _beta_one, double _beta_two, unsigned long _iteration, double _epsilon, double _alpha) {
    beta_one = _beta_one;
    beta_two = _beta_two;
    iteration = _iteration;
    epsilon = _epsilon;
    alpha = _alpha;
  }

  __host__ __device__ double operator()(const double& m, const double& nu) {
    return alpha * ((m / (1 - pow(beta_one, iteration))) / (sqrt(nu / (1 - pow(beta_two, iteration))) + epsilon));
  }
};


struct output_functor {
  double dropout_rate;

  output_functor(double _dropout_rate){
    dropout_rate = _dropout_rate;
  }

  __host__ __device__ double operator()(const double& inputValue, const double& weight) const {
    return inputValue * (weight * (1.0 - dropout_rate));
  }
};


/**
 * vectorのサイズ確保のためだけに用いるNeuronのデフォルトコンストラクタ
 * @return Neuronのインスタンス
 */
Neuron::Neuron() {}

/**
 * Neuronのコンストラクタ
 * @param num_input 入力ニューロン数（入力データ数）
 * @param dropout_rate Dropout率
 * @return Neuronのインスタンス
 */
Neuron::Neuron(const unsigned long num_input, const vector<double> &weight,
               const vector<double> &m, const vector<double> &nu,
               const unsigned long iteration, const double bias, const int activation_type,
               const double dropout_rate) {
  this->num_input = num_input; // このニューロンへの入力数（前の層のニューロン数）
  this->activation_type = activation_type;
  this->dropout_rate = dropout_rate;
  random_device rnd; // 非決定的乱数生成器
  mt19937 mt; // メルセンヌ・ツイスタ
  mt.seed(rnd());
  uniform_real_distribution<double> real_rnd(0.0, 1.0);

  if (bias != 0.0) this->bias = bias;
  else this->bias = real_rnd(mt); // バイアスを乱数で設定

  // Adamの各パラメータについて，学習済みのものが渡されていればセットし，そうでなければ0.0で初期化
  if (iteration != 0) this->iteration = iteration;
  else this->iteration = 0;


  if (m.size() > 0) this->d_m = thrust::device_vector<double>(m);
  else this->d_m = thrust::device_vector<double>(num_input, 0.0);

  if (nu.size() > 0) this->d_nu = thrust::device_vector<double>(nu);
  else this->d_nu = thrust::device_vector<double>(num_input, 0.0);

  // 結合荷重が渡されていればそれをセットし，無ければ乱数で初期化
  if (weight.size() > 0) this->d_inputWeights = thrust::device_vector<double>(weight);
  else {
    this->d_inputWeights.resize(num_input);
    for (int i = 0; i < this->num_input; ++i) this->d_inputWeights[i] = real_rnd(mt);
  }

  d_adam_result = thrust::device_vector<double>(num_input);
  d_output_result = thrust::device_vector<double>(num_input);
  d_learn_output_result = thrust::device_vector<double>(num_input);

  h_inputWeights.resize(num_input);
  h_m.resize(num_input);
  h_nu.resize(num_input);
}

/**
 * 受け取った0.0以上1.0未満の乱数値からdropout_maskを設定する
 * @param random_value 0.0以上1.0未満の乱数値
 */
void Neuron::dropout(const double random_value) {
  if (random_value < dropout_rate) this->dropout_mask = 0.0;
  else this->dropout_mask = 1.0;
}

/**
 * dropout_maskが1.0であれば，Adamを用いてニューロンの結合荷重を学習し，確率的勾配降下でバイアスを更新する
 * @param delta 損失関数を偏微分したもの（これに一つ前の層の出力データを掛けて傾きを得る）
 * @param inputValues 一つ前の層の出力データ
 */
void Neuron::learn(const double delta, const vector<double> &inputValues) {
  this->delta = delta;

  // Adamを用いて重み付けを学習する
  if (this->dropout_mask == 1.0) {
    this->iteration += 1;

    d_inputValues = inputValues;

    // transform m inputValues using learn_m_functor
    thrust::transform(d_m.begin(), d_m.end(),
                      d_inputValues.begin(), d_m.begin(), learn_m_functor(delta, beta_one));

    thrust::transform(d_nu.begin(), d_nu.end(),
                      d_inputValues.begin(), d_nu.begin(), learn_nu_functor(delta, beta_two));

    thrust::transform(d_m.begin(), d_m.end(), d_nu.begin(), d_adam_result.begin(),
                      learn_functor(beta_one, beta_two, iteration, epsilon, alpha));
    thrust::transform(d_inputWeights.begin(), d_inputWeights.end(), d_adam_result.begin(),
                      d_inputWeights.begin(), thrust::minus<double>());

    thrust::copy(d_m.begin(), d_m.end(), h_m.begin());
    thrust::copy(d_nu.begin(), d_nu.end(), h_nu.begin());
    thrust::copy(d_inputWeights.begin(), d_inputWeights.end(), h_inputWeights.begin());

    // 確率的勾配降下でバイアスを更新
    this->bias -= (this->alpha * this->delta) - (this->alpha * this->rambda * this->bias);
  }
}

/**
 * ニューロンの出力メソッド．バイアスや重み付けにdropout_ratioを掛けて処理する
 * @param inputValues 一つ前の層の出力データ
 * @return ニューロンの出力値（活性化関数より得られた値）
 */
double Neuron::output(const vector<double> &inputValues) {
  double sum = this->bias * (1.0 - this->dropout_rate);

  d_inputValues = inputValues;

  thrust::transform(d_inputValues.begin(), d_inputValues.end(),
                    d_inputWeights.begin(), d_output_result.begin(),
                    output_functor(dropout_rate));
  sum += thrust::reduce(d_output_result.begin(), d_output_result.end());

  double activated;
  if (activation_type == 0) activated = activation_identity(sum);
  else if (activation_type == 1) activated = activation_sigmoid(sum);
  else if (activation_type == 2) activated = activation_tanh(sum);
  else activated = activation_relu(sum);

  return activated;
}

/**
 * ニューロンの出力を得て，それにdropout_maskを掛ける
 * @param inputValues ニューロンの入力データ
 * @return ニューロンの出力
 */
double Neuron::learn_output(const vector<double> &inputValues) {
  // 入力側の細胞出力の重み付き和をとる
  double sum = this->bias;

  d_inputValues = inputValues;

  thrust::transform(d_inputValues.begin(), d_inputValues.end(),
                    d_inputWeights.begin(), d_learn_output_result.begin(),
                    thrust::multiplies<double>());
  sum += thrust::reduce(d_learn_output_result.begin(), d_learn_output_result.end());

  // 得られた重み付き和を活性化関数に入れて出力を得る
  double activated;
  if (activation_type == 0) activated = activation_identity(sum);
  else if (activation_type == 1) activated = activation_sigmoid(sum);
  else if (activation_type == 2) activated = activation_tanh(sum);
  else activated = activation_relu(sum);

  return activated * this->dropout_mask;
}

/**
 * 活性化関数：恒等写像
 * @param x 入力
 * @return 計算結果
 */
double Neuron::activation_identity(const double x) {
  return x;
}

/**
 * 活性化関数 : シグモイド関数
 * @param x 入力
 * @return 計算結果
 */
double Neuron::activation_sigmoid(const double x) {
  return 1.0 / (1.0 + pow(M_E, -x));
}

/**
 * 活性化関数 : tanh
 * @param x 入力
 * @return 計算結果
 */
double Neuron::activation_tanh(const double x) {
  return tanh(x);
}

/**
 * 活性化関数 : ランプ関数（ReLU）
 * @param x 入力
 * @return 計算結果
 */
double Neuron::activation_relu(const double x) {
  return max(0.0, x);
}

/**
 * このニューロンの指定された入力インデックスの結合荷重を返す
 * @param i 入力インデックス
 * @return 結合荷重
 */
double Neuron::getInputWeightIndexOf(const int i) {
  return this->h_inputWeights[i];
}

/**
 * 現在の修正量を返す
 * @return 修正量
 */
double Neuron::getDelta() {
  return this->delta;
}

/**
 * このニューロンの閾値を返す
 */
double Neuron::getBias() {
  return this->bias;
}

double Neuron::getMIndexOf(const int i) {
  return this->h_m[i];
}

double Neuron::getNuIndexOf(const int i) {
  return this->h_nu[i];
}

unsigned long Neuron::getIteration() {
  return this->iteration;
}

/**
 * このニューロンの結合荷重を文字列でまとめて返す
 * @return このニューロンの結合荷重をまとめた文字列
 */
string Neuron::toString() {
  stringstream ss;
  ss << "weight : ";
  for (int neuron = 0; neuron < num_input; ++neuron)
    ss << h_inputWeights[neuron] << " , ";

  string output = ss.str();
  return output;
}
